#include "hip/hip_runtime.h"
#include<cstdio>
#include <torch/extension.h>
#include <c10/cuda/CUDAGuard.h>
#define BLOCK_SIZE 256
#define DIVUP(n) (int)ceil((float)n / BLOCK_SIZE)

#define CHECK_CUDA(x) \
  TORCH_CHECK(x.device().is_cuda(), #x, " must be a CUDAtensor ")

__global__ void add_kernel_impl(const float* a, const float* b, float* res, uint32_t n){
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (idx < n){
        res[idx] = a[idx] + b[idx];
    }
}

void add_launcher(const float* a, const float* b, float* res, uint32_t n){
    dim3 threadSize(BLOCK_SIZE);
    dim3 blockSize((uint32_t)ceil((float) n / BLOCK_SIZE));
    add_kernel_impl<<<blockSize, threadSize>>>(a, b, res, n);
}

at::Tensor add_gpu(const at::Tensor &a_tensor, const at::Tensor &b_tensor){
    
    // The line below must to be used, or cuda report errors.
    at::cuda::OptionalCUDAGuard device_guard(a_tensor.device());
    CHECK_CUDA(a_tensor);
    CHECK_CUDA(b_tensor);

    at::Tensor a_contig = a_tensor.contiguous();
    at::Tensor b_contig = b_tensor.contiguous();
    at::Tensor result = torch::zeros(a_contig.sizes(), a_contig.options());
    //at::Tensor result = a_contig.clone() also the same.

    uint32_t n = 1;
    for (size_t i = 0; i < a_contig.sizes().size(); ++i){
      n *= a_tensor.size(i);
    }

    const float* a = a_tensor.data_ptr<float>();
    const float* b = b_tensor.data_ptr<float>();
    float* res = result.data_ptr<float>();

    add_launcher(a, b, res, n);

    return result;
}